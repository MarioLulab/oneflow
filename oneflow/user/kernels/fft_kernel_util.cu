#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#if 1
#include <hip/hip_runtime.h>

#if CUDA_VERSION >= 11000

#include "cufft_plan_cache.h"
#include "oneflow/user/kernels/fft_kernel_util.h"

namespace oneflow {

#if 1
namespace {

template<typename IN, typename OUT>
__global__ void convert_complex_to_real(IN* dst, const OUT* src, size_t n) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    dst[2 * i] = src[i].x;
    dst[2 * i + 1] = src[i].y;
  };
}

double _fft_normalization_scale(const int32_t frame_length) {
  return static_cast<double>(1.0 / std::sqrt(frame_length));
}

template<typename FFTTYPE>
__global__ void fft_apply_normalization(FFTTYPE* dst, const double normalization_scale, size_t n,
                                        bool IsNormalized) {
  if (!IsNormalized) { return; }
  CUDA_1D_KERNEL_LOOP(i, n) {
    dst[i].x *= normalization_scale;
    dst[i].y *= normalization_scale;
  };
}

// TODO(yzm):support doublesided
template<typename FFTTYPE>
__global__ void convert_doublesided(const FFTTYPE* src, FFTTYPE* dst, size_t len, size_t n) {
  size_t fact_len = 2 * len - 2;
  CUDA_1D_KERNEL_LOOP(i, n) {
    int index_x = i / fact_len;
    int index_y = i % fact_len;
    if (index_y == 0) {
      dst[i] = src[index_x * len];
    } else if (index_y == len - 1) {
      dst[i] = src[(index_x + 1) * len - 1];
    } else if (index_y < len - 1 && index_y > 0) {
      dst[i] = src[index_x * len + index_y];
    } else {
      auto index = (index_x + 2) * len - index_y - 2;
      dst[i].x = src[index].x;
      dst[i].y = -src[index].y;
    }
  }
}

}  // namespace
#endif

#if 0
template<typename IN, typename OUT>
class StftGpuKernel final : public user_op::OpKernel {
 public:
  StftGpuKernel() = default;
  ~StftGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* input = ctx->Tensor4ArgNameAndIndex("input", 0);
    user_op::Tensor* output = ctx->Tensor4ArgNameAndIndex("output", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const bool normalized = ctx->Attr<bool>("normalized");
    const bool onesided = ctx->Attr<bool>("onesided");
    const bool return_complex = ctx->Attr<bool>("return_complex");

    const ShapeView& input_shape = input->shape_view();
    const ShapeView& output_shape = output->shape_view();

    const Stride& input_stride = input->stride();
    const int out_elem_cnt =
        return_complex ? output->shape_view().elem_cnt() : output->shape_view().elem_cnt() / 2;

    const IN* data_in = input->dptr<IN>();
    IN* data_out = output->mut_dptr<IN>();
    OUT* out_tmp_buffer = reinterpret_cast<OUT*>(tmp_buffer->mut_dptr<char>());

    int32_t ndim = 1;
    int32_t n_frames = static_cast<int32_t>(input_shape.At(1));
    int32_t fft_size = static_cast<int32_t>(input_shape.At(2));
    const Stride& in_stride = {input_stride.at(2), input_stride.at(1)};
    const Stride& out_stride = {1, fft_size / 2 + 1};
    const Shape& in_shape = {fft_size, n_frames};
    const Shape& out_shape = in_shape;
    int32_t batch = n_frames;
    int32_t rank[1] = {fft_size};
    CuFFtParams params(ndim, rank, in_stride, out_stride, in_shape, out_shape, batch);
    CuFFtConfig<IN, OUT> config(params);

    int32_t in_offset = input_stride.at(0);
    int32_t out_offset = n_frames * (fft_size / 2 + 1);
    int32_t signal_groups_count = static_cast<int32_t>(input_shape.At(0));
    for (int32_t i = 0; i < signal_groups_count; i++) {
      config.excute_plan(data_in + i * in_offset, out_tmp_buffer + i * out_offset);
    }

    if (!onesided) {
      size_t last_dim_length = fft_size / 2 + 1;
      OUT* doublesided_tmp_buffer =
          reinterpret_cast<OUT*>(tmp_buffer->mut_dptr<char>()) + out_elem_cnt;
      convert_doublesided<<<BlocksNum4ThreadsNum(out_elem_cnt), kCudaThreadsNumPerBlock, 0,
                            ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          out_tmp_buffer, doublesided_tmp_buffer, last_dim_length, out_elem_cnt);
      out_tmp_buffer = doublesided_tmp_buffer;
    }

    const double normalization_scale = _fft_normalization_scale(input_shape.back());
    fft_apply_normalization<<<BlocksNum4ThreadsNum(out_elem_cnt), kCudaThreadsNumPerBlock, 0,
                              ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
        out_tmp_buffer, normalization_scale, out_elem_cnt, normalized);

    if (!return_complex) {
      convert_complex_to_real<<<BlocksNum4ThreadsNum(out_elem_cnt), kCudaThreadsNumPerBlock, 0,
                                ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          data_out, out_tmp_buffer, out_elem_cnt);
    } else {
      // TODO(yzm):support return_complex after oneflow supports complex numbers
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};
#define REGISTER_STFT_GPU_KERNEL(intype, outtype)                                           \
  REGISTER_USER_KERNEL("stft")                                                              \
      .SetCreateFn<StftGpuKernel<intype, outtype>>()                                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                      \
                       && (user_op::HobDataType("input", 0) == GetDataType<intype>::value)) \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                                   \
        const Shape& output_shape = ctx->InputShape("output", 0);                           \
        const bool return_complex = ctx->Attr<bool>("return_complex");                      \
        const bool onesided = ctx->Attr<bool>("onesided");                                  \
        int64_t output_elem_cnt =                                                           \
            return_complex ? output_shape.elem_cnt() : output_shape.elem_cnt() / 2;         \
        const int64_t output_bytes = GetCudaAlignedSize(output_elem_cnt * sizeof(outtype)); \
        return onesided ? output_bytes : 2 * output_bytes;                                  \
      });

REGISTER_STFT_GPU_KERNEL(float, hipfftComplex)
REGISTER_STFT_GPU_KERNEL(double, hipfftDoubleComplex)
#endif

// Execute a general fft operation (can be c2c, onesided r2c or onesided c2r)
template<typename IN, typename OUT>
static void DoFFT(IN* in, OUT* out,
                  const Stride& in_stride, const Shape& in_shape, 
                  std::vector<int64_t>& out_sizes, std::vector<int64_t>& fft_dims, bool forward)
{
  const int64_t ndim = in_stride.size();
  const int64_t fft_ndim = fft_dims.size();
  const int64_t batch_dims = ndim - fft_ndim;


  // Permute dimensions to make batch dims come first, and this maximizes data locality
  std::vector<int64_t> dim_permute(ndim);
  std::iota(dim_permute.begin(), dim_permute.end(), int64_t(0));
  std::vector<bool> is_transformed_dim(ndim, false);
  for (const auto& dim : fft_dims){
    is_transformed_dim[dim] = true;
  }

  auto batch_end = std::partition(dim_permute.begin(), dim_permute.end(),
                                        [&](int64_t d) {return !is_transformed_dim[d];});
  std::sort(dim_permute.begin(), batch_end,
            [&](int64_t a, int64_t b) { return in_stride[a] > in_stride[b]; });
  std::copy(fft_dims.begin(), fft_dims.end(), batch_end);
  // permute
  std::vector<int64_t> working_in_stride(dim_permute.size(), 0);
  std::vector<int64_t> working_in_shape(dim_permute.size(), 0);
  FOR_RANGE(int64_t, i, 0, dim_permute.size()){
    working_in_shape[i] = in_shape[dim_permute[i]];
    working_in_stride[i] = in_stride[dim_permute[i]];
  }

  std::vector<int64_t> batched_sizes(fft_ndim + 1);
  int64_t batch = 1;
  FOR_RANGE(int64_t, i, 0, working_in_shape.size() - fft_ndim){
    batch *= working_in_shape[i];
  }
  // input = input.reshape(batched_sizes)
  // maybe method:
  // `1
  // 1. judge if compact
  // 2. if compact, no need to be contiguous
  // 3. change working_in_shape and working_in_stride 
  // `2
  // 1. judge if compact
  // 2. if compact, just change working_in_shape and working_in_stride
  // 3. if not compact, construct `MemcpyFactory` like reshape kernel

}

template<typename T>
class FftC2CKernelUtil<DeviceType::kCUDA, T>{
  static void FftC2CForward(ep::Stream* stream, const T* data_in, T* data_out, T* tmp_buffer,  
                            const Shape& input_shape, const Shape& output_shape, const Shape& tmp_buffer_shape,
                            const Stride& input_stride, const Stride& output_stride, const Stride& tmp_buffer_stride, 
                            bool forward,
                            const std::vector<int64_t>& dims, fft_norm_mode normalization){
    std::vector<int64_t> sorted_dims(dims.begin(), dims.end());
    Shape working_tensor_shape = input_shape;
    Stride working_tensor_stride = input_stride;
    T* working_data_ptr = data_in;

    while (true){
      std::sort(sorted_dims.begin(), sorted_dims.end(), 
            [&](int64_t a, int64_t b) { return working_tensor_stride[a] > working_tensor_stride[b];});

      size_t cur_fft_ndims = std::min(static_cast<size_t>(max_rank), sorted_dims.size());
      std::vector<int64_t> cur_fft_dims(sorted_dims.end() - cur_fft_ndims, sorted_dims.end());

      // DoFFT

      // after DoFFT
      sorted_dims.resize(sorted_dims.size() - cur_fft_ndims);

      if (sorted_dims.empty()){
        break;
      }

      if (working_data_ptr == data_in){
          working_data_ptr = data_out;
          // working_tensor_shape = 
      }
    }

    // input -> c2c -> output -> c2c -> tmp_buffer

  }
};

template<typename IN, typename OUT>
struct FftR2CKernelUtil<DeviceType::kCUDA, IN, OUT> {
  static void FftR2CForward(ep::Stream* stream, const IN* data_in, OUT* data_out,
                            const Shape& input_shape, const Shape& output_shape,
                            const Stride& input_stride, const Stride& output_stride, bool forward,
                            const std::vector<int64_t>& dims, fft_norm_mode normalization){
    // TO-DO:
    UNIMPLEMENTED();
  }
};

template<typename IN, typename OUT>
struct FftC2RKernelUtil<DeviceType::kCUDA, IN, OUT> {
  static void FftC2RForward(ep::Stream* stream, const IN* data_in, OUT* data_out,
                            const Shape& input_shape, const Shape& output_shape,
                            const Stride& input_stride, const Stride& output_stride,
                            int64_t last_dim_size, const std::vector<int64_t>& dims,
                            fft_norm_mode normalization){
    // TO-DO:
    UNIMPLEMENTED();
  }
};

}  // namespace oneflow

#endif

#endif