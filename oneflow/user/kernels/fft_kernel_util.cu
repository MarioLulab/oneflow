#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/framework/user_op_tensor.h"
#include "oneflow/user/kernels/to_contiguous_kernel.h"
#if 1
#include <hip/hip_runtime.h>

#if CUDA_VERSION >= 11000

#include "cufft_plan_cache.h"
#include "oneflow/user/kernels/fft_kernel_util.h"

namespace oneflow {

#if 1
namespace {

template<typename IN, typename OUT>
__global__ void convert_complex_to_real(IN* dst, const OUT* src, size_t n) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    dst[2 * i] = src[i].x;
    dst[2 * i + 1] = src[i].y;
  };
}

double _fft_normalization_scale(const int32_t frame_length) {
  return static_cast<double>(1.0 / std::sqrt(frame_length));
}

template<typename FFTTYPE>
__global__ void fft_apply_normalization(FFTTYPE* dst, const double normalization_scale, size_t n,
                                        bool IsNormalized) {
  if (!IsNormalized) { return; }
  CUDA_1D_KERNEL_LOOP(i, n) {
    dst[i].x *= normalization_scale;
    dst[i].y *= normalization_scale;
  };
}

// TODO(yzm):support doublesided
template<typename FFTTYPE>
__global__ void convert_doublesided(const FFTTYPE* src, FFTTYPE* dst, size_t len, size_t n) {
  size_t fact_len = 2 * len - 2;
  CUDA_1D_KERNEL_LOOP(i, n) {
    int index_x = i / fact_len;
    int index_y = i % fact_len;
    if (index_y == 0) {
      dst[i] = src[index_x * len];
    } else if (index_y == len - 1) {
      dst[i] = src[(index_x + 1) * len - 1];
    } else if (index_y < len - 1 && index_y > 0) {
      dst[i] = src[index_x * len + index_y];
    } else {
      auto index = (index_x + 2) * len - index_y - 2;
      dst[i].x = src[index].x;
      dst[i].y = -src[index].y;
    }
  }
}

bool isCompact(const std::vector<int64_t>& strides, const std::vector<int64_t>& shape){
  if (strides.size() != shape.size()){
    return false;
  }
  Shape shape_(shape);
  Stride stride_(shape_);
  FOR_RANGE(int64_t, i, 0, strides.size()){
    if (strides[i] != stride_[i]){
      return false;
    }
  }
  return true;
}

}  // namespace
#endif

#if 0
template<typename IN, typename OUT>
class StftGpuKernel final : public user_op::OpKernel {
 public:
  StftGpuKernel() = default;
  ~StftGpuKernel() = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* input = ctx->Tensor4ArgNameAndIndex("input", 0);
    user_op::Tensor* output = ctx->Tensor4ArgNameAndIndex("output", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const bool normalized = ctx->Attr<bool>("normalized");
    const bool onesided = ctx->Attr<bool>("onesided");
    const bool return_complex = ctx->Attr<bool>("return_complex");

    const ShapeView& input_shape = input->shape_view();
    const ShapeView& output_shape = output->shape_view();

    const Stride& input_stride = input->stride();
    const int out_elem_cnt =
        return_complex ? output->shape_view().elem_cnt() : output->shape_view().elem_cnt() / 2;

    const IN* data_in = input->dptr<IN>();
    IN* data_out = output->mut_dptr<IN>();
    OUT* out_tmp_buffer = reinterpret_cast<OUT*>(tmp_buffer->mut_dptr<char>());

    int32_t ndim = 1;
    int32_t n_frames = static_cast<int32_t>(input_shape.At(1));
    int32_t fft_size = static_cast<int32_t>(input_shape.At(2));
    const Stride& in_stride = {input_stride.at(2), input_stride.at(1)};
    const Stride& out_stride = {1, fft_size / 2 + 1};
    const Shape& in_shape = {fft_size, n_frames};
    const Shape& out_shape = in_shape;
    int32_t batch = n_frames;
    int32_t rank[1] = {fft_size};
    CuFFtParams params(ndim, rank, in_stride, out_stride, in_shape, out_shape, batch);
    CuFFtConfig<IN, OUT> config(params);

    int32_t in_offset = input_stride.at(0);
    int32_t out_offset = n_frames * (fft_size / 2 + 1);
    int32_t signal_groups_count = static_cast<int32_t>(input_shape.At(0));
    for (int32_t i = 0; i < signal_groups_count; i++) {
      config.excute_plan(data_in + i * in_offset, out_tmp_buffer + i * out_offset);
    }

    if (!onesided) {
      size_t last_dim_length = fft_size / 2 + 1;
      OUT* doublesided_tmp_buffer =
          reinterpret_cast<OUT*>(tmp_buffer->mut_dptr<char>()) + out_elem_cnt;
      convert_doublesided<<<BlocksNum4ThreadsNum(out_elem_cnt), kCudaThreadsNumPerBlock, 0,
                            ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          out_tmp_buffer, doublesided_tmp_buffer, last_dim_length, out_elem_cnt);
      out_tmp_buffer = doublesided_tmp_buffer;
    }

    const double normalization_scale = _fft_normalization_scale(input_shape.back());
    fft_apply_normalization<<<BlocksNum4ThreadsNum(out_elem_cnt), kCudaThreadsNumPerBlock, 0,
                              ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
        out_tmp_buffer, normalization_scale, out_elem_cnt, normalized);

    if (!return_complex) {
      convert_complex_to_real<<<BlocksNum4ThreadsNum(out_elem_cnt), kCudaThreadsNumPerBlock, 0,
                                ctx->stream()->As<ep::CudaStream>()->hip_stream()>>>(
          data_out, out_tmp_buffer, out_elem_cnt);
    } else {
      // TODO(yzm):support return_complex after oneflow supports complex numbers
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};
#define REGISTER_STFT_GPU_KERNEL(intype, outtype)                                           \
  REGISTER_USER_KERNEL("stft")                                                              \
      .SetCreateFn<StftGpuKernel<intype, outtype>>()                                        \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA)                      \
                       && (user_op::HobDataType("input", 0) == GetDataType<intype>::value)) \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                                   \
        const Shape& output_shape = ctx->InputShape("output", 0);                           \
        const bool return_complex = ctx->Attr<bool>("return_complex");                      \
        const bool onesided = ctx->Attr<bool>("onesided");                                  \
        int64_t output_elem_cnt =                                                           \
            return_complex ? output_shape.elem_cnt() : output_shape.elem_cnt() / 2;         \
        const int64_t output_bytes = GetCudaAlignedSize(output_elem_cnt * sizeof(outtype)); \
        return onesided ? output_bytes : 2 * output_bytes;                                  \
      });

REGISTER_STFT_GPU_KERNEL(float, hipfftComplex)
REGISTER_STFT_GPU_KERNEL(double, hipfftDoubleComplex)
#endif
#if 0
// Execute a general fft operation (can be c2c, onesided r2c or onesided c2r)
template<typename IN, typename OUT>
static void DoFFT(ep::Stream* stream, IN* in, OUT* out,
                  const Stride& in_stride, const Shape& in_shape, 
                  std::vector<int64_t>& out_sizes, std::vector<int64_t>& fft_dims, bool forward)
{
  const int64_t ndim = in_stride.size();
  const int64_t fft_ndim = fft_dims.size();
  const int64_t batch_dims = ndim - fft_ndim;


  // Permute dimensions to make batch dims come first, and this maximizes data locality
  std::vector<int64_t> dim_permute(ndim);
  std::iota(dim_permute.begin(), dim_permute.end(), int64_t(0));
  std::vector<bool> is_transformed_dim(ndim, false);
  for (const auto& dim : fft_dims){
    is_transformed_dim[dim] = true;
  }

  auto batch_end = std::partition(dim_permute.begin(), dim_permute.end(),
                                        [&](int64_t d) {return !is_transformed_dim[d];});
  std::sort(dim_permute.begin(), batch_end,
            [&](int64_t a, int64_t b) { return in_stride[a] > in_stride[b]; });
  std::copy(fft_dims.begin(), fft_dims.end(), batch_end);

  // permute
  std::vector<int64_t> working_in_stride(dim_permute.size(), 0);
  std::vector<int64_t> working_in_shape(dim_permute.size(), 0);
  FOR_RANGE(int64_t, i, 0, dim_permute.size()){
    working_in_shape[i] = in_shape[dim_permute[i]];
    working_in_stride[i] = in_stride[dim_permute[i]];
  }

  std::vector<int64_t> batched_sizes(fft_ndim + 1);
  int64_t batch = 1;
  FOR_RANGE(int64_t, i, 0, working_in_shape.size() - fft_ndim){
    batch *= working_in_shape[i];
  }
  // input = input.reshape(batched_sizes)
  // maybe method:
  // `1
  // 1. judge if compact
  // 2. if compact, no need to be contiguous, else be contiguous
  // 3. change working_in_shape and working_in_stride 
  // `2
  // 1. judge if compact
  // 2. if compact, just change working_in_shape and working_in_stride
  // 3. if not compact, construct `MemcpyFactory` like reshape kernel
  if (!isCompact(/*strides=*/working_in_stride, /*shape=*/working_in_shape)){
    ToContiguousUtil<DeviceType::kCUDA, IN>(stream, )
  }
  else{

  }


}
#endif

template<typename T, typename FCT_TYPE>
class FftC2CKernelUtil<DeviceType::kCUDA, T, FCT_TYPE>{
  static void FftC2CForward(ep::Stream* stream, const T* data_in, T* data_out, 
                            const Shape& input_shape, const Shape& output_shape,
                            const Stride& input_stride, const Stride& output_stride, 
                            bool forward, const std::vector<int64_t>& dims, FCT_TYPE normalization,
                            DataType real_type){
    CuFFTParams params(input_shape, output_shape, input_stride, output_stride, 
                      dims.size(), forward, CUFFT_EXCUTETYPE::C2C, real_type);
    CuFFTConfig config(params);
    auto& plan = config.plan();
    CUFFT_CHECK(hipfftSetStream(plan, stream->As<ep::CudaStream>()->hip_stream()));
    void* workspace{};
    OF_CUDA_CHECK(hipMalloc(&workspace, config.workspace_size()));
    CUFFT_CHECK(hipfftSetWorkArea(plan, workspace));

    config.excute((void*)data_in, (void*)data_out, forward);
    OF_CUDA_CHECK(hipFree(workspace));
  }
};


template<typename IN, typename OUT>
struct FftR2CKernelUtil<DeviceType::kCUDA, IN, OUT> {
  static void FftR2CForward(ep::Stream* stream, const IN* data_in, OUT* data_out,
                            const Shape& input_shape, const Shape& output_shape,
                            const Stride& input_stride, const Stride& output_stride, bool forward,
                            const std::vector<int64_t>& dims, IN normalization){
    // TO-DO:
    UNIMPLEMENTED();
  }
};

template<typename IN, typename OUT>
struct FftC2RKernelUtil<DeviceType::kCUDA, IN, OUT> {
  static void FftC2RForward(ep::Stream* stream, const IN* data_in, OUT* data_out,
                            const Shape& input_shape, const Shape& output_shape,
                            const Stride& input_stride, const Stride& output_stride,
                            int64_t last_dim_size, const std::vector<int64_t>& dims,
                            OUT normalization){
    // TO-DO:
    UNIMPLEMENTED();
  }
};

template struct FftC2CKernelUtil<DeviceType::kCUDA, hipComplex, float>;
template struct FftC2CKernelUtil<DeviceType::kCUDA, hipDoubleComplex, double>;

}  // namespace oneflow

#endif

#endif