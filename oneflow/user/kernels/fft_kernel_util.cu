#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/framework/user_op_tensor.h"
#include "oneflow/user/kernels/to_contiguous_kernel.h"
#if 1
#include <hip/hip_runtime.h>

#if CUDA_VERSION >= 11000

#include "cufft_plan_cache.h"
#include "oneflow/user/kernels/fft_kernel_util.h"

namespace oneflow {

namespace {

template<typename IN, typename OUT>
__global__ void convert_complex_to_real(IN* dst, const OUT* src, size_t n) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    dst[2 * i] = src[i].x;
    dst[2 * i + 1] = src[i].y;
  };
}

double _fft_normalization_scale(const int32_t frame_length) {
  return static_cast<double>(1.0 / std::sqrt(frame_length));
}

template<typename FFTTYPE>
__global__ void fft_apply_normalization(FFTTYPE* dst, const double normalization_scale, size_t n,
                                        bool IsNormalized) {
  if (!IsNormalized) { return; }
  CUDA_1D_KERNEL_LOOP(i, n) {
    dst[i].x *= normalization_scale;
    dst[i].y *= normalization_scale;
  };
}

// TODO(yzm):support doublesided
template<typename FFTTYPE>
__global__ void convert_doublesided(const FFTTYPE* src, FFTTYPE* dst, size_t len, size_t n) {
  size_t fact_len = 2 * len - 2;
  CUDA_1D_KERNEL_LOOP(i, n) {
    int index_x = i / fact_len;
    int index_y = i % fact_len;
    if (index_y == 0) {
      dst[i] = src[index_x * len];
    } else if (index_y == len - 1) {
      dst[i] = src[(index_x + 1) * len - 1];
    } else if (index_y < len - 1 && index_y > 0) {
      dst[i] = src[index_x * len + index_y];
    } else {
      auto index = (index_x + 2) * len - index_y - 2;
      dst[i].x = src[index].x;
      dst[i].y = -src[index].y;
    }
  }
}

template<int NDIM>
struct FillConjSymmetricParams {
  int64_t last_dim;
  int64_t elem_count;
  oneflow::NdIndexStrideOffsetHelper<int64_t, NDIM> helper;
  int64_t last_dim_size;
  int64_t last_dim_half;

  FillConjSymmetricParams() = default;
  FillConjSymmetricParams(const Shape& shape, const Stride& strides, 
                          int64_t last_dim_, int64_t elemcnt) : last_dim(last_dim_), 
                          elem_count(elemcnt), helper(strides.data(), NDIM)
  {
    assert(strides.size() == shape.size());
    assert(NDIM == strides.size());
    last_dim_size = shape[last_dim];
    last_dim_half = last_dim_size / 2;
  }
};

}  // namespace

template<typename T, int NDIM>
__global__ void _conj_symmetry_cuda(T* data_out, FillConjSymmetricParams<NDIM> param) {
  CUDA_1D_KERNEL_LOOP_T(int64_t, offset, param.elem_count){
    int64_t indices[NDIM];
    param.helper.OffsetToNdIndex(offset, indices, NDIM);
    if (indices[param.last_dim] <= param.last_dim_half){
      continue;
    }
    int64_t cur_last_dim_index = indices[param.last_dim];
    // get symmetric
    indices[param.last_dim] = param.last_dim_size - cur_last_dim_index;
    int64_t symmetric_offset = param.helper.NdIndexToOffset(indices, NDIM);

    // conj
    data_out[offset] = T{data_out[symmetric_offset].x, - data_out[symmetric_offset].y};
  }

}

template<typename T>
struct FillConjSymmetryUtil<DeviceType::kCPU, T>{
  static void FillConjSymmetryForward(ep::Stream* stream, T* data_out, const Shape& shape, const Stride& strides,
                                      const int64_t last_dim, int64_t elem_count){
    switch (shape.size()) {
      case 1:{
        FillConjSymmetricParams<1> param(shape, strides, last_dim, elem_count);
        _conj_symmetry_cuda<T, 1><<<BlocksNum4ThreadsNum(elem_count), kCudaThreadsNumPerBlock, 0,
                             stream->As<ep::CudaStream>()->hip_stream()>>>(
                                    data_out, param);
        };
        break;
      case 2:{
        FillConjSymmetricParams<2> param(shape, strides, last_dim, elem_count);
        _conj_symmetry_cuda<T, 2><<<BlocksNum4ThreadsNum(elem_count), kCudaThreadsNumPerBlock, 0,
                             stream->As<ep::CudaStream>()->hip_stream()>>>(
                                    data_out, param);
        };
        break;
      case 3:{
        FillConjSymmetricParams<3> param(shape, strides, last_dim, elem_count);
        _conj_symmetry_cuda<T, 3><<<BlocksNum4ThreadsNum(elem_count), kCudaThreadsNumPerBlock, 0,
                             stream->As<ep::CudaStream>()->hip_stream()>>>(
                                    data_out, param);
        };
        break;
      case 4:{
        FillConjSymmetricParams<4> param(shape, strides, last_dim, elem_count);
        _conj_symmetry_cuda<T, 4><<<BlocksNum4ThreadsNum(elem_count), kCudaThreadsNumPerBlock, 0,
                             stream->As<ep::CudaStream>()->hip_stream()>>>(
                                    data_out, param);
        };
        break;
      case 4:{
        FillConjSymmetricParams<4> param(shape, strides, last_dim, elem_count);
        _conj_symmetry_cuda<T, 4><<<BlocksNum4ThreadsNum(elem_count), kCudaThreadsNumPerBlock, 0,
                             stream->As<ep::CudaStream>()->hip_stream()>>>(
                                    data_out, param);
        };
        break;
      case 5:{
        FillConjSymmetricParams<5> param(shape, strides, last_dim, elem_count);
        _conj_symmetry_cuda<T, 5><<<BlocksNum4ThreadsNum(elem_count), kCudaThreadsNumPerBlock, 0,
                             stream->As<ep::CudaStream>()->hip_stream()>>>(
                                    data_out, param);
        };
        break;
      case 6:{
        FillConjSymmetricParams<6> param(shape, strides, last_dim, elem_count);
        _conj_symmetry_cuda<T, 6><<<BlocksNum4ThreadsNum(elem_count), kCudaThreadsNumPerBlock, 0,
                             stream->As<ep::CudaStream>()->hip_stream()>>>(
                                    data_out, param);
        };
        break;
      case 7:{
        FillConjSymmetricParams<7> param(shape, strides, last_dim, elem_count);
        _conj_symmetry_cuda<T, 7><<<BlocksNum4ThreadsNum(elem_count), kCudaThreadsNumPerBlock, 0,
                             stream->As<ep::CudaStream>()->hip_stream()>>>(
                                    data_out, param);
        };
        break;
      case 8:{
        FillConjSymmetricParams<8> param(shape, strides, last_dim, elem_count);
        _conj_symmetry_cuda<T, 8><<<BlocksNum4ThreadsNum(elem_count), kCudaThreadsNumPerBlock, 0,
                             stream->As<ep::CudaStream>()->hip_stream()>>>(
                                    data_out, param);
        };
        break;
      case 9:{
        FillConjSymmetricParams<9> param(shape, strides, last_dim, elem_count);
        _conj_symmetry_cuda<T, 9><<<BlocksNum4ThreadsNum(elem_count), kCudaThreadsNumPerBlock, 0,
                             stream->As<ep::CudaStream>()->hip_stream()>>>(
                                    data_out, param);
        };
        break;
      case 10:{
        FillConjSymmetricParams<10> param(shape, strides, last_dim, elem_count);
        _conj_symmetry_cuda<T, 10><<<BlocksNum4ThreadsNum(elem_count), kCudaThreadsNumPerBlock, 0,
                             stream->As<ep::CudaStream>()->hip_stream()>>>(
                                    data_out, param);
        };
        break;
      case 11:{
        FillConjSymmetricParams<11> param(shape, strides, last_dim, elem_count);
        _conj_symmetry_cuda<T, 11><<<BlocksNum4ThreadsNum(elem_count), kCudaThreadsNumPerBlock, 0,
                             stream->As<ep::CudaStream>()->hip_stream()>>>(
                                    data_out, param);
        };
        break;
      case 12:{
        FillConjSymmetricParams<12> param(shape, strides, last_dim, elem_count);
        _conj_symmetry_cuda<T, 12><<<BlocksNum4ThreadsNum(elem_count), kCudaThreadsNumPerBlock, 0,
                             stream->As<ep::CudaStream>()->hip_stream()>>>(
                                    data_out, param);
        };
        break;
      default: UNIMPLEMENTED(); break;
    }
  }
};

template<typename T, typename FCT_TYPE>
class FftC2CKernelUtil<DeviceType::kCUDA, T, FCT_TYPE>{
  static void FftC2CForward(ep::Stream* stream, const T* data_in, T* data_out, 
                            const Shape& input_shape, const Shape& output_shape,
                            const Stride& input_stride, const Stride& output_stride, 
                            bool forward, const std::vector<int64_t>& dims, FCT_TYPE normalization,
                            DataType real_type){
    CuFFTParams params(input_shape, output_shape, input_stride, output_stride, 
                      dims.size(), CUFFT_EXCUTETYPE::C2C, real_type);
    CuFFTConfig config(params);
    auto& plan = config.plan();
    OF_CUFFT_CHECK(hipfftSetStream(plan, stream->As<ep::CudaStream>()->hip_stream()));
    void* workspace{};
    OF_CUDA_CHECK(hipMalloc(&workspace, config.workspace_size()));
    OF_CUFFT_CHECK(hipfftSetWorkArea(plan, workspace));

    config.excute((void*)data_in, (void*)data_out, forward);
    OF_CUDA_CHECK(hipFree(workspace));
  }
};


template<typename IN, typename OUT>
struct FftR2CKernelUtil<DeviceType::kCUDA, IN, OUT> {
  static void FftR2CForward(ep::Stream* stream, const IN* data_in, OUT* data_out,
                            const Shape& input_shape, const Shape& output_shape,
                            const Stride& input_stride, const Stride& output_stride, bool forward,
                            const std::vector<int64_t>& dims, IN normalization, DataType real_type){
    CuFFTParams params(input_shape, output_shape, input_stride, output_stride, 
                      dims.size(), CUFFT_EXCUTETYPE::R2C, real_type);
    CuFFTConfig config(params);
    auto& plan = config.plan();
    OF_CUFFT_CHECK(hipfftSetStream(plan, stream->As<ep::CudaStream>()->hip_stream()));
    void* workspace{};
    OF_CUDA_CHECK(hipMalloc(&workspace, config.workspace_size()));
    OF_CUFFT_CHECK(hipfftSetWorkArea(plan, workspace));

    config.excute((void*)data_in, (void*)data_out, forward);
    OF_CUDA_CHECK(hipFree(workspace));    
  }
};

template<typename IN, typename OUT>
struct FftC2RKernelUtil<DeviceType::kCUDA, IN, OUT> {
  static void FftC2RForward(ep::Stream* stream, const IN* data_in, OUT* data_out,
                            const Shape& input_shape, const Shape& output_shape,
                            const Stride& input_stride, const Stride& output_stride,
                            int64_t last_dim_size, const std::vector<int64_t>& dims,
                            OUT normalization, DataType real_type){
    // TO-DO:
    UNIMPLEMENTED();
  }
};

template struct FftC2CKernelUtil<DeviceType::kCUDA, hipComplex, float>;
template struct FftC2CKernelUtil<DeviceType::kCUDA, hipDoubleComplex, double>;

}  // namespace oneflow

#endif

#endif